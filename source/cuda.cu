#include "hip/hip_runtime.h"
/*
This file contains some function for GPU operation
*/
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../include/ParSecureML_noMPI.h"
/*
Malloc GPU memory
*/
void MallocD(float *&gpu_a, int size){
    hipError_t cudaStat;
    cudaStat = hipMalloc((void**)&gpu_a, sizeof(*gpu_a)*size);
    if(cudaStat != hipSuccess){
        cout << "Malloc failed:" << hipGetErrorString(cudaStat)<< endl;
        exit(0);
    }
}
/*
Transfer data from host to device
*/
void CopyHtoD(float *gpu_a, float *a, int size){
    hipError_t cudaStat;
    cudaStat = hipMemcpy(gpu_a, a, sizeof(*a)*size, hipMemcpyHostToDevice);
    if(cudaStat != hipSuccess){
        cout << "Error code:" << cudaStat << endl;
        cout << "CopyHtoD failed." << endl;
        exit(0);
    }
}

/*
Transfer data from device to host
*/
void CopyDtoH(float *&a, float *&gpu_a, int size){
    hipError_t cudaStat;
    cudaStat = hipMemcpy(a, gpu_a, sizeof(*a)*size, hipMemcpyDeviceToHost);
    if(cudaStat != hipSuccess){
        cout << "Error code:" << cudaStat << endl;
        cout << "CopyDtoH failed." << endl;
        exit(0);
    }
}

/*
Matrix multiplication for Support class
*/
void Support_noMPI::GPU_Mul(){
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat = hipblasCreate(&handle);
    stat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "CUBLAS create failed." << endl;
        exit(0);
    }
    float alpha = 1;
    float b = 0;
    
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha, GPU_U, row1, GPU_V, row2, &b, GPU_Z, row1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "Cublas sgemm failed." << endl;
        exit(0);
    }
}

void ReleaseGPU(float *A){
    hipFree(A);
}

/*
Triplet sum function
sum = fac1+fac2+fac3
*/
__global__ void cudaTripletSum(float *sum, float *fac1, float *fac2, float *fac3,  int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = fac1[cur] + fac2[cur] + fac3[cur];
    sum[cur] = tmp;
}

/*
sum = A + B
*/
__global__ void cudaSum(float *A, float *B, float *sum, int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = A[cur]+B[cur];
    sum[cur] = tmp;
}

/*
min = A - B
*/
__global__ void cudaMinus(float *A, float *B, float *min, int size){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= size) return;
    float tmp = A[cur]-B[cur];
    min[cur] = tmp;
}

/*
Triplet multiplication
if server1:
    result = A*F + E*B + Z
if server2:
    result = (A-E)*F + E*B + Z
*/
void Triplet_noMPI::cudaTripletMul(int flag){
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipError_t cudaStat;
    stat = hipblasCreate(&handle);
    stat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "CUBLAS create failed." << endl;
        exit(0);
    }
    float alpha1 = 1;
    float alpha2 = 1;
    float b = 0;
    while(flag1 == 0){
        continue;
    }
    if(flag == 0){
        while(flag2 == 0){
            continue;
        }
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha1, GPU_A, row1, GPU_F, row2, &b, fac1, row1);
        if(stat != HIPBLAS_STATUS_SUCCESS){
            cout << "Cublas sgemm failed." << endl;
            exit(0);
        }
    }
    else if(flag == 1){
        cudaMinus<<<row1*col1/THREADNUM+1, THREADNUM>>>(GPU_A, GPU_E, GPU_D, row1*col1);
        while(flag2 == 0){
            continue;
        }
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha1, GPU_D, row1, GPU_F, row2, &b, fac1, row1);
        if(stat != HIPBLAS_STATUS_SUCCESS){
            cout << "Cublas sgemm failed." << endl;
            exit(0);
        }
    }
    while(flag3 == 0){
        continue;
    }
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row1, col2, col1, &alpha2, GPU_E, row1, GPU_B, row2, &b, fac2, row1);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        cout << "Cublas sgemm failed." << endl;
        exit(0);
    }
    hipblasDestroy(handle);
    cudaTripletSum<<<row1*col1/THREADNUM+THREADNUM, THREADNUM>>>(GPU_C, fac1, fac2, GPU_Z, row1*col2);
    cudaStat = hipGetLastError();
    if(cudaStat != hipSuccess){
        cout << "Kernel launch failed." << endl;
        exit(0);
    }
}

/*
Convolution for SecureML on GPU
*/
__global__ void cudaConv(int flag, float *GPU_A, float *GPU_B, float *GPU_C, float *GPU_E, float *GPU_F, float *GPU_Z, int row1, int col1, int row2, int col2, int o_row, int o_col, int num){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int cur = bid*blockDim.x+tid;
    if(cur >= num*o_row*o_col) return;
    int num_cur = cur/(o_row*o_col);
    int row_cur = cur%(o_row*o_col)/o_col;
    int col_cur = cur%(o_row*o_col)%o_col;
    float tem = 0;
    for(int i = 0; i < row2; i++){
        for(int j = 0; j < col2; j++){
            tem += flag*GPU_E[num_cur*o_row*o_col*row2*col2+row_cur*o_col*row2*col2+col_cur*row2*col2+i*col2*j]*GPU_F[i*col2+j] + GPU_A[num_cur*row1*col1+(row_cur+i)*row1+col_cur+j] * GPU_F[i*col2+j] + GPU_E[num_cur*o_row*o_col*row2*col2+row_cur*o_col*row2*col2+col_cur*row2*col2+i*col2*j] * GPU_B[i*col2+j] + GPU_Z[i*col2+j];
        }
    }
    GPU_C[num_cur*o_row*o_col+row_cur*o_col+col_cur] = tem;
}
void ConvTriplet_noMPI::GPU_OP(int flag){
    cudaConv<<<num*o_row*o_col/THREADNUM+1, THREADNUM>>>(flag, GPU_A, GPU_B, GPU_C, GPU_E, GPU_F, GPU_Z, row1, col1, row2, col2, o_row, o_row, num);
}
